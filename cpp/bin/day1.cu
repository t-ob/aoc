// 1000 lines
// max 52 chars

// Kernel definition

#include <hip/hip_runtime.h>
#include <string>
#include <iostream>

// in[1024][64], out[1024]

static const int ROWS = 1024;
static const int COLS = 64;
static const int THREADS_PER_BLOCK = 256;

__global__ void part1Kernel(uint8_t *in, uint8_t *out, unsigned int numRows)
{
    auto i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= numRows) return;

    uint8_t nums[2];

    auto k = 0;

    for (auto j = 0; j < 64; ++j) {
        auto c = in[COLS * i + j];
        if (c < '0' || c > '9') continue;

        if (k == 0) {
            nums[0] = c - '0';
            nums[1] = c - '0';
            k = 1;
        } else {
            nums[1] = c - '0';
        }
    }

    out[i] = 10 * nums[0] + nums[1];
}

__global__ void part2Kernel(uint8_t *in, int8_t *out, unsigned int numRows)
{
    auto i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= numRows) return;

    uint8_t nums[2];

    auto k = 0;
    auto l = in[COLS * i + COLS - 1];

    for (auto j = 0; j < 63; ++j) {
        auto c = in[COLS * i + j];
        if (c < '0' || c > '9') continue;

        if (k == 0) {
            nums[0] = c - '0';
            nums[1] = c - '0';
            k = 1;
        } else {
            nums[1] = c - '0';
        }
    }

    out[i] = 10 * nums[0] + nums[1];
}

__global__ void sumKernel(uint8_t *input, uint32_t *partialSums, int size) {
    __shared__ uint32_t shared[THREADS_PER_BLOCK];
    auto tid = threadIdx.x;
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    shared[tid] = (i < size) ? input[i] : 0;
    __syncthreads();

    for (auto s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared[tid] += shared[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) partialSums[blockIdx.x] = shared[0];
}



int main() {
    auto in = (uint8_t *) malloc(ROWS * COLS);
    auto out = (uint8_t *) malloc(ROWS);
    std::fill(in, in + (ROWS * COLS), static_cast<uint8_t>(0));
    std::fill(out, out + ROWS, static_cast<uint8_t>(0));

    auto i = 0;

    std::string line;
    while (std::getline(std::cin, line)) {
        auto j = 0;
        for (const auto c : line) {
            in[COLS * i + j] = c;
            ++j;
        }
        ++i;
    }

    uint8_t* dIn;
    hipMalloc(&dIn, sizeof(uint8_t) * ROWS * COLS);
    uint8_t* dOut;
    hipMalloc(&dOut, sizeof(uint8_t) * ROWS);


    hipMemcpy(dIn, in, sizeof(uint8_t) * ROWS * COLS, hipMemcpyHostToDevice);

    auto blocks = (ROWS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    part1Kernel<<<blocks, THREADS_PER_BLOCK>>>(dIn, dOut, i);

    uint32_t *dPartialSums;
    hipMalloc((void**)&dPartialSums, blocks * sizeof(uint32_t));

    sumKernel<<<blocks, THREADS_PER_BLOCK>>>(dOut, dPartialSums, i);

    uint32_t partialSums[blocks];
    hipMemcpy(partialSums, dPartialSums, blocks * sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint32_t totalSum = 0;
    for (auto b = 0; b < blocks; b++) {
        totalSum += partialSums[b];
    }

    std::cout << std::to_string(totalSum) << std::endl;

    hipFree(dIn);
    hipFree(dOut);
    hipFree(dPartialSums);

    return 0;
}