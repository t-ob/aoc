// 1000 lines
// max 52 chars

// Kernel definition

#include <hip/hip_runtime.h>
#include <cstring>
#include <string>
#include <iostream>

// in[1024][64], out[1024]

static const int ROWS = 1024;
static const int MAX_CHARS = 64;
static const int MAX_SEQ_LEN = 8;

static const int BLOCKS = 32;
static const int ROW_THREADS = 32;
static const int COL_THREADS = 32;


__global__ void part2Kernel(const char *in, const char *seqs, const uint8_t *values, uint32_t *out, unsigned int numRows, unsigned int numSeqs)
{
    __shared__ int32_t sharedIndexes[32][32];
    __shared__ int32_t sharedMaxIndexes[32][32];
    __shared__ uint32_t sharedArgMin[32][32];
    __shared__ uint32_t sharedArgMax[32][32];
    __shared__ uint32_t sharedSums[32];

    // Initialise shared memory
    sharedIndexes[threadIdx.x][threadIdx.y] = -1;
    sharedMaxIndexes[threadIdx.x][threadIdx.y] = -1;
    sharedArgMin[threadIdx.x][threadIdx.y] = threadIdx.y;
    sharedArgMax[threadIdx.x][threadIdx.y] = threadIdx.y;
    if (threadIdx.y == 0) sharedSums[threadIdx.x] = 0;
    __syncthreads();

    auto rowIdx = blockDim.x * blockIdx.x + threadIdx.x;
    auto seqIdx = threadIdx.y;

    // Nothing to do for threads outside rowIdx or seqIdx
    if (rowIdx >= numRows) return;
    if (seqIdx >= numSeqs) return;


    // Compute first and last occurrences (if any) of each sequence
    {
        auto i = 0;
        while (i < MAX_CHARS) {
            auto j = 0;
            while ((i + j < MAX_CHARS) && in[rowIdx * MAX_CHARS + i + j] != 0 && in[rowIdx * MAX_CHARS + i + j] == seqs[seqIdx * MAX_SEQ_LEN + j]) {
                ++j;
            }

            // Check if entire token has been found
            if (seqs[seqIdx * MAX_SEQ_LEN + j] == 0) {
                sharedIndexes[threadIdx.x][seqIdx] = i;
                break;
            }

            ++i;
        }
    }

    {
        auto i = MAX_CHARS - 1;
        while (i >= 0) {
            auto j = 0;
            while ((i + j < MAX_CHARS) && in[rowIdx * MAX_CHARS + i + j] != 0 && in[rowIdx * MAX_CHARS + i + j] == seqs[seqIdx * MAX_SEQ_LEN + j]) {
                ++j;
            }

            // Check if entire token has been found
            if (seqs[seqIdx * MAX_SEQ_LEN + j] == 0) {
                sharedMaxIndexes[threadIdx.x][seqIdx] = i;
                break;
            }

            --i;
        }
    }

    __syncthreads();

    // Compute argmins and argmaxes
    for (auto s = COL_THREADS >> 1; s > 0; s >>= 1) {
        if (seqIdx < s) {
            auto leftArgMinIdx = sharedArgMin[threadIdx.x][seqIdx];
            auto rightArgMinIdx = sharedArgMin[threadIdx.x][seqIdx + s];

            auto leftArgMinCandidate = sharedIndexes[threadIdx.x][leftArgMinIdx];
            auto rightArgMinCandidate = sharedIndexes[threadIdx.x][rightArgMinIdx];

            if ((leftArgMinCandidate < 0 && rightArgMinCandidate < 0) || (leftArgMinCandidate >= 0 && rightArgMinCandidate < 0))  {
                sharedArgMin[threadIdx.x][seqIdx] = leftArgMinIdx;
            } else if (leftArgMinCandidate < 0) {
                sharedArgMin[threadIdx.x][seqIdx] = rightArgMinIdx;
            } else if (leftArgMinCandidate < rightArgMinCandidate) {
                sharedArgMin[threadIdx.x][seqIdx] = leftArgMinIdx;
            } else {
                sharedArgMin[threadIdx.x][seqIdx] = rightArgMinIdx;
            }

            auto leftArgMaxIdx = sharedArgMax[threadIdx.x][seqIdx];
            auto rightArgMaxIdx = sharedArgMax[threadIdx.x][seqIdx + s];

            auto leftArgMaxCandidate = sharedMaxIndexes[threadIdx.x][leftArgMaxIdx];
            auto rightArgMaxCandidate = sharedMaxIndexes[threadIdx.x][rightArgMaxIdx];

            if ((leftArgMaxCandidate < 0 && rightArgMaxCandidate < 0) || (leftArgMaxCandidate >= 0 && rightArgMaxCandidate < 0)) {
                sharedArgMax[threadIdx.x][seqIdx] = leftArgMaxIdx;
            } else if (leftArgMaxCandidate < 0) {
                sharedArgMax[threadIdx.x][seqIdx] = rightArgMaxIdx;
            } else if (leftArgMaxCandidate < rightArgMaxCandidate) {
                sharedArgMax[threadIdx.x][seqIdx] = rightArgMaxIdx;
            } else {
                sharedArgMax[threadIdx.x][seqIdx] = leftArgMaxIdx;
            }
        }
        __syncthreads();
    }

    // Compute partial sum for block
    if (seqIdx == 0) {
        auto a = 10 * values[sharedArgMin[threadIdx.x][0]] + values[sharedArgMax[threadIdx.x][0]];
        sharedSums[threadIdx.x] = a;
        __syncthreads();

        // Compute sum of computed values in block
        for (auto s = ROW_THREADS >> 1; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                sharedSums[threadIdx.x] += sharedSums[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) out[blockIdx.x] = sharedSums[0];
    }
}


int main() {
    // Input buffer
    auto in = (char *) malloc(ROWS * MAX_CHARS);
    std::fill(in, in + (ROWS * MAX_CHARS), static_cast<uint8_t>(0));

    auto i = 0;
    std::string line;
    while (std::getline(std::cin, line)) {
        auto j = 0;
        for (const auto c : line) {
            in[MAX_CHARS * i + j] = c;
            ++j;
        }
        ++i;
    }

    char* dIn;
    hipMalloc(&dIn, sizeof(uint8_t) * ROWS * MAX_CHARS);
    hipMemcpy(dIn, in, sizeof(uint8_t) * ROWS * MAX_CHARS, hipMemcpyHostToDevice);

    // All possible strings to search
    const int numStrings = 19;
    const int maxLen = 8;
    char hostStrings[numStrings][maxLen];

    strcpy(hostStrings[0], "0");
    strcpy(hostStrings[1], "1");
    strcpy(hostStrings[2], "2");
    strcpy(hostStrings[3], "3");
    strcpy(hostStrings[4], "4");
    strcpy(hostStrings[5], "5");
    strcpy(hostStrings[6], "6");
    strcpy(hostStrings[7], "7");
    strcpy(hostStrings[8], "8");
    strcpy(hostStrings[9], "9");
    strcpy(hostStrings[10], "one");
    strcpy(hostStrings[11], "two");
    strcpy(hostStrings[12], "three");
    strcpy(hostStrings[13], "four");
    strcpy(hostStrings[14], "five");
    strcpy(hostStrings[15], "six");
    strcpy(hostStrings[16], "seven");
    strcpy(hostStrings[17], "eight");
    strcpy(hostStrings[18], "nine");

    char* dStrings;
    hipMalloc((void**)&dStrings, numStrings * maxLen * sizeof(char));
    hipMemcpy(dStrings, hostStrings, numStrings * maxLen * sizeof(char), hipMemcpyHostToDevice);

    // Lookup table for tokens by their index
    uint8_t hValues[numStrings] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    uint8_t *dValues;
    hipMalloc(&dValues, numStrings * sizeof(uint8_t));
    hipMemcpy(dValues, hValues, numStrings * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Memory for each block to write its partial sum to
    uint32_t* dOutPart1;
    hipMalloc(&dOutPart1, BLOCKS * sizeof(uint32_t));

    uint32_t* dOutPart2;
    hipMalloc(&dOutPart2, BLOCKS * sizeof(uint32_t));

    // Launch kernels
    dim3 threadsPerBlock(ROW_THREADS, COL_THREADS);
    part2Kernel<<<BLOCKS, threadsPerBlock>>>(dIn, dStrings, dValues, dOutPart1, i, 10);
    part2Kernel<<<BLOCKS, threadsPerBlock>>>(dIn, dStrings, dValues, dOutPart2, i, 19);
    hipDeviceSynchronize();

    uint32_t outPart1[BLOCKS];
    hipMemcpy(outPart1, dOutPart1, BLOCKS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint32_t outPart2[BLOCKS];
    hipMemcpy(outPart2, dOutPart2, BLOCKS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint32_t totalPart1 = 0;
    for (auto m : outPart1) {
        totalPart1 += m;
    }

    uint32_t totalPart2 = 0;
    for (auto m : outPart2) {
        totalPart2 += m;
    }

    std::cout << totalPart1 << std::endl;
    std::cout << totalPart2 << std::endl;

    hipFree(dIn);
    hipFree(dStrings);
    hipFree(dValues);
    hipFree(dOutPart1);

    return 0;
}