#include "hip/hip_runtime.h"
// max games = 6

#include <string>
#include <iostream>
#include <sstream>

static const int BLOCKS = 4;
static const int GAME_THREADS_PER_BLOCK = 32;
static const int ROUND_THREADS_PER_BLOCK = 8;
static const int RGB_THREADS_PER_BLOCK = 4;

__global__ void kernel(const uint8_t* games, const uint8_t* rounds, const uint8_t *rgb, uint32_t *outPart1, uint32_t *outPart2) {
    __shared__ bool part1Data[32][8][4];
    __shared__ uint32_t part2Data[32][8][4];
    __shared__ bool part1RoundIsSolvable[32][8];
    __shared__ uint32_t part1GameIdSum[32];
    __shared__ uint32_t part2PowerSum[32];

    part1Data[threadIdx.x][threadIdx.y][threadIdx.z] = false;

    __syncthreads();

    auto gameIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gameIdx >= 100) return;

    if (threadIdx.y >= rounds[gameIdx]) return;

    if (threadIdx.z >= 3) return;

    auto target = rgb[threadIdx.z];
    auto index = gameIdx * (6 * 3) + threadIdx.y * 3 + threadIdx.z;
    auto value = games[index];

    part1Data[threadIdx.x][threadIdx.y][threadIdx.z] = value <= target;
    part2Data[threadIdx.x][threadIdx.y][threadIdx.z] = value;

    __syncthreads();

    if (threadIdx.z == 0) {
        auto outcome = part1Data[threadIdx.x][threadIdx.y];
        part1RoundIsSolvable[threadIdx.x][threadIdx.y] = outcome[0] && outcome[1] && outcome[2];

        __syncthreads();

        for (auto s = 8 >> 1; s > 0; s >>= 1) {
            if (threadIdx.y < s && threadIdx.y + s < rounds[gameIdx]) {
                part1RoundIsSolvable[threadIdx.x][threadIdx.y] = part1RoundIsSolvable[threadIdx.x][threadIdx.y] && part1RoundIsSolvable[threadIdx.x][threadIdx.y + s];
            }
            __syncthreads();
        }
    }

    if (threadIdx.y == 0 && threadIdx.z == 0) {
        part1GameIdSum[threadIdx.x] = static_cast<uint32_t>(part1RoundIsSolvable[threadIdx.x][0]) * (gameIdx + 1);

        __syncthreads();

        for (auto s = 32 >> 1; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                part1GameIdSum[threadIdx.x] += part1GameIdSum[threadIdx.x + s];
            }
            __syncthreads();
        }
    }

    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) outPart1[blockIdx.x] = part1GameIdSum[0];

    __syncthreads();

    for (auto s = 8 >> 1; s > 0; s >>= 1) {
        if (threadIdx.y < s) {
            part2Data[threadIdx.x][threadIdx.y][threadIdx.z] = max(part2Data[threadIdx.x][threadIdx.y][threadIdx.z], part2Data[threadIdx.x][threadIdx.y + s][threadIdx.z]);
        }
        __syncthreads();
    }

    if (threadIdx.y == 0 && threadIdx.z == 0) {
        part2PowerSum[threadIdx.x] = (part2Data[threadIdx.x][0][0] * part2Data[threadIdx.x][0][1] * part2Data[threadIdx.x][0][2]);
        __syncthreads();

        for (auto s = 32 >> 1; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                part2PowerSum[threadIdx.x] += part2PowerSum[threadIdx.x + s];
            }
            __syncthreads();
        }
    }
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) outPart2[blockIdx.x] = part2PowerSum[0];
}

int main() {
    uint8_t games[100][6][3] = {};
    uint8_t rounds[100] = {};

    auto numLines = 0;
    std::string line;
    while (std::getline(std::cin, line)) {
        size_t pos = line.find(':');
        line = line.substr(pos + 1);

        std::istringstream ss(line);
        std::string record;
        int row = 0;

        while (std::getline(ss, record, ';') && row < 6) {
            std::istringstream recordStream(record);
            std::string word;
            int count;

            for (int i = 0; i < 3; ++i) {
                games[numLines][row][i] = 0;
            }

            while (recordStream >> count >> word) {
                if (word.ends_with(',')) word.pop_back();
                if (word == "red" || word == "red,") {
                    games[numLines][row][0] = count;
                } else if (word == "green") {
                    games[numLines][row][1] = count;
                } else if (word == "blue") {
                    games[numLines][row][2] = count;
                }
            }

            row++;
        }

        rounds[numLines] = row;
        ++numLines;
    }

    uint8_t* dGames;
    hipMalloc(&dGames, sizeof(games));
    hipMemcpy(dGames, games, sizeof(games), hipMemcpyHostToDevice);

    uint8_t* dRounds;
    hipMalloc(&dRounds, sizeof(rounds));
    hipMemcpy(dRounds, rounds, sizeof(rounds), hipMemcpyHostToDevice);

    uint8_t rgb[3] = {12, 13, 14};
    uint8_t* dRgb;
    hipMalloc(&dRgb, sizeof(rgb));
    hipMemcpy(dRgb, rgb, sizeof(rgb), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(GAME_THREADS_PER_BLOCK, ROUND_THREADS_PER_BLOCK, RGB_THREADS_PER_BLOCK);

    uint32_t* dOutPart1;
    hipMalloc(&dOutPart1, sizeof(uint32_t) * BLOCKS);
    uint32_t* dOutPart2;
    hipMalloc(&dOutPart2, sizeof(uint32_t) * BLOCKS);
    kernel<<<BLOCKS, threadsPerBlock>>>(dGames, dRounds, dRgb, dOutPart1, dOutPart2);
    hipDeviceSynchronize();
    uint32_t outPart1[BLOCKS];
    hipMemcpy(outPart1, dOutPart1, sizeof(uint32_t) * BLOCKS, hipMemcpyDeviceToHost);
    uint32_t outPart2[BLOCKS];
    hipMemcpy(outPart2, dOutPart2, sizeof(uint32_t) * BLOCKS, hipMemcpyDeviceToHost);

    uint32_t totalPart1 = 0;
    for (const auto s : outPart1) totalPart1 += s;

    uint32_t totalPart2 = 0;
    for (const auto s : outPart2) totalPart2 += s;

    std::cout << totalPart1 << std::endl;
    std::cout << totalPart2 << std::endl;

    return 0;
}