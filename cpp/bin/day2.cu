#include "hip/hip_runtime.h"
// max games = 6

#include <string>
#include <iostream>
#include <sstream>

static const int MAX_GAMES = 100;
static const int MAX_ROUNDS = 6;
static const int CHANNELS = 3;
static const int BLOCKS = 4;
static const int GAME_THREADS_PER_BLOCK = 32;
static const int ROUND_THREADS_PER_BLOCK = 8;
static const int RGB_THREADS_PER_BLOCK = 4;

__global__ void
kernel(const uint8_t *games, const uint8_t *rounds, const uint8_t *rgb, uint32_t *outPart1, uint32_t *outPart2) {
    __shared__ bool part1Data[GAME_THREADS_PER_BLOCK][ROUND_THREADS_PER_BLOCK][RGB_THREADS_PER_BLOCK];
    __shared__ uint32_t part2Data[GAME_THREADS_PER_BLOCK][ROUND_THREADS_PER_BLOCK][RGB_THREADS_PER_BLOCK];
    __shared__ bool part1RoundIsSolvable[GAME_THREADS_PER_BLOCK][ROUND_THREADS_PER_BLOCK];
    __shared__ uint32_t part1GameIdSum[GAME_THREADS_PER_BLOCK];
    __shared__ uint32_t part2PowerSum[GAME_THREADS_PER_BLOCK];

    // Will answer "does the channel value at a given round for a given game satisfy the target"
    part1Data[threadIdx.x][threadIdx.y][threadIdx.z] = false;

    __syncthreads();

    // Don't do work on threads where none is required
    auto gameIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gameIdx >= MAX_GAMES) return;
    if (threadIdx.y >= rounds[gameIdx]) return;
    if (threadIdx.z >= CHANNELS) return;

    auto target = rgb[threadIdx.z];
    // Do the uint8_t[100][6][3] to *uint8_t index arithmetic
    auto index = gameIdx * (MAX_ROUNDS * CHANNELS) + threadIdx.y * CHANNELS + threadIdx.z;
    auto value = games[index];

    part1Data[threadIdx.x][threadIdx.y][threadIdx.z] = value <= target;

    // We'll reduce this up the y-axis later for the power components of part 2
    part2Data[threadIdx.x][threadIdx.y][threadIdx.z] = value;

    __syncthreads();

    // Determine which games are solvable for part 1
    if (threadIdx.z == 0) {
        auto outcome = part1Data[threadIdx.x][threadIdx.y];
        part1RoundIsSolvable[threadIdx.x][threadIdx.y] = outcome[0] && outcome[1] && outcome[2];

        __syncthreads();

        for (auto s = ROUND_THREADS_PER_BLOCK >> 1; s > 0; s >>= 1) {
            if (threadIdx.y < s && threadIdx.y + s < rounds[gameIdx]) {
                part1RoundIsSolvable[threadIdx.x][threadIdx.y] = part1RoundIsSolvable[threadIdx.x][threadIdx.y] &&
                                                                 part1RoundIsSolvable[threadIdx.x][threadIdx.y + s];
            }
            __syncthreads();
        }
    }

    // Add the IDs of the solvable games
    if (threadIdx.y == 0 && threadIdx.z == 0) {
        part1GameIdSum[threadIdx.x] = static_cast<uint32_t>(part1RoundIsSolvable[threadIdx.x][0]) * (gameIdx + 1);

        __syncthreads();

        for (auto s = GAME_THREADS_PER_BLOCK >> 1; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                part1GameIdSum[threadIdx.x] += part1GameIdSum[threadIdx.x + s];
            }
            __syncthreads();
        }
    }

    // Move the final output at position 0 to the correct part of the output
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) outPart1[blockIdx.x] = part1GameIdSum[0];

    __syncthreads();

    // Build up the max across each channel for all rounds in a game
    for (auto s = ROUND_THREADS_PER_BLOCK >> 1; s > 0; s >>= 1) {
        if (threadIdx.y < s) {
            part2Data[threadIdx.x][threadIdx.y][threadIdx.z] = max(part2Data[threadIdx.x][threadIdx.y][threadIdx.z],
                                                                   part2Data[threadIdx.x][threadIdx.y +
                                                                                          s][threadIdx.z]);
        }
        __syncthreads();
    }

    // Compute their power and sum them
    if (threadIdx.y == 0 && threadIdx.z == 0) {
        part2PowerSum[threadIdx.x] = (part2Data[threadIdx.x][0][0] * part2Data[threadIdx.x][0][1] *
                                      part2Data[threadIdx.x][0][2]);
        __syncthreads();

        for (auto s = GAME_THREADS_PER_BLOCK >> 1; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                part2PowerSum[threadIdx.x] += part2PowerSum[threadIdx.x + s];
            }
            __syncthreads();
        }
    }

    // Move the final output at position 0 to the correct part of the output
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) outPart2[blockIdx.x] = part2PowerSum[0];
}

int main() {
    uint8_t games[MAX_GAMES][MAX_ROUNDS][CHANNELS] = {};
    uint8_t rounds[MAX_GAMES] = {};

    // Parse input data
    auto game = 0;
    std::string line;
    while (std::getline(std::cin, line)) {
        size_t pos = line.find(':');
        line = line.substr(pos + 1);

        std::istringstream ss(line);
        std::string record;
        int round = 0;

        while (std::getline(ss, record, ';') && round < MAX_ROUNDS) {
            std::istringstream recordStream(record);
            std::string word;
            int count;

            for (int channel = 0; channel < CHANNELS; ++channel) {
                games[game][round][channel] = 0;
            }

            while (recordStream >> count >> word) {
                if (word.ends_with(',')) word.pop_back();
                if (word == "red" || word == "red,") {
                    games[game][round][0] = count;
                } else if (word == "green") {
                    games[game][round][1] = count;
                } else if (word == "blue") {
                    games[game][round][2] = count;
                }
            }

            round++;
        }

        rounds[game] = round;
        ++game;
    }

    // Move parsed input to device
    uint8_t *dGames;
    hipMalloc(&dGames, sizeof(games));
    hipMemcpy(dGames, games, sizeof(games), hipMemcpyHostToDevice);

    // Move rounds per game data to device
    uint8_t *dRounds;
    hipMalloc(&dRounds, sizeof(rounds));
    hipMemcpy(dRounds, rounds, sizeof(rounds), hipMemcpyHostToDevice);

    // Move part 1 targets to device
    uint8_t rgb[3] = {12, 13, 14};
    uint8_t *dRgb;
    hipMalloc(&dRgb, sizeof(rgb));
    hipMemcpy(dRgb, rgb, sizeof(rgb), hipMemcpyHostToDevice);

    // Allocate memory for output for both parts
    uint32_t *dOutPart1;
    hipMalloc(&dOutPart1, sizeof(uint32_t) * BLOCKS);
    uint32_t *dOutPart2;
    hipMalloc(&dOutPart2, sizeof(uint32_t) * BLOCKS);

    dim3 threadsPerBlock(GAME_THREADS_PER_BLOCK, ROUND_THREADS_PER_BLOCK, RGB_THREADS_PER_BLOCK);

    kernel<<<BLOCKS, threadsPerBlock>>>(dGames, dRounds, dRgb, dOutPart1, dOutPart2);
    hipDeviceSynchronize();

    // Retrieve outputs
    uint32_t outPart1[BLOCKS];
    hipMemcpy(outPart1, dOutPart1, sizeof(uint32_t) * BLOCKS, hipMemcpyDeviceToHost);
    uint32_t outPart2[BLOCKS];
    hipMemcpy(outPart2, dOutPart2, sizeof(uint32_t) * BLOCKS, hipMemcpyDeviceToHost);

    uint32_t totalPart1 = 0;
    for (const auto s: outPart1) totalPart1 += s;

    uint32_t totalPart2 = 0;
    for (const auto s: outPart2) totalPart2 += s;

    std::cout << totalPart1 << std::endl;
    std::cout << totalPart2 << std::endl;

    return 0;
}